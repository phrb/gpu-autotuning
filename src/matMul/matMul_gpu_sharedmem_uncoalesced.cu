
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define Tile_Width 16

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void matMul(float* Pd, float* Md, float* Nd, int Width) {
  __shared__ float Mds[Tile_Width][Tile_Width];
  __shared__ float Nds[Tile_Width][Tile_Width];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Identify the row and column of the M element to work on
  int Col = blockIdx.x * Tile_Width + tx;
  int Row = blockIdx.y * Tile_Width + ty;

  float Pvalue = 0;
  // Loop over the N and P tiles required to compute the M element
  for (int m = 0; m < Width/Tile_Width; ++m) {
    // Coolaborative loading of N and P tiles into shared memory
    Mds[tx][ty] = Md[Col*Width + (m*Tile_Width + ty)];
    Nds[tx][ty] = Nd[Row + (m*Tile_Width + tx)*Width];
    __syncthreads();

    for (int k = 0; k < Tile_Width; ++k)
      Pvalue += Mds[tx][k] * Nds[k][ty];
    __syncthreads();
  }
  Pd[Col * Width + Row] = Pvalue;
}


// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
  for (int k = 0; k < size; ++k) {
     data[k] = (float)drand48();
  }
}

int main(int argc, char* argv[])
{

  if (argc != 4) {
    fprintf(stderr, "Syntax: %s <matrix size Width> < Block_size> <CacheConfL1> \n", argv[0]);
    return EXIT_FAILURE;
  }

  int Width = atoi(argv[1]);
  int BlockSize = atoi(argv[2]);
  int devId = 0;
  int CacheConfL1 = atoi(argv[3]);

  checkCuda( hipSetDevice(devId) );
  hipDeviceReset();

  // allocate host memory for matrices M and N
  printf("Allocate host memory for matrices M and N...\n");
  float* M = (float*) malloc(Width * Width * sizeof(float));
  float* N = (float*) malloc(Width * Width * sizeof(float));
  float* P = (float*) malloc(Width * Width * sizeof(float));

  // set seed for drand48()
  srand48(42);

  // initialize host matrices
  printf("Initialize host matrices...\n");
  randomInit(M, Width*Width);
  randomInit(N, Width*Width);

  // allocate device matrices (linearized)
  printf("Allocate device matrices (linearized)...\n");
  float* Md = NULL; 
  float* Nd = NULL;
  float* Pd = NULL;
  checkCuda( hipMalloc((void**) &Md, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Nd, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Pd, Width * Width * sizeof(float)) );

  // copy host memory to device
  checkCuda( hipMemcpy(Md, M, Width*Width*sizeof(float), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(Nd, N, Width*Width*sizeof(float), hipMemcpyHostToDevice) );

  // execute the kernel
  printf("Execute the kernel...\n");

  if (CacheConfL1 == 1){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferShared);
  }
  else if (CacheConfL1 == 2){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferEqual);
  }
  else if (CacheConfL1 == 3){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferL1);
  }
  else {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferNone);
  }

  int GridSize = (Width + Tile_Width-1) / Tile_Width;
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(Tile_Width, Tile_Width);

  hipProfilerStart();
  matMul<<< gridDim, blockDim >>>(Pd, Md, Nd, Width);
  hipProfilerStop();

  // copy result from device to host
  checkCuda( hipMemcpy( P, Pd, Width * Width * sizeof(float),hipMemcpyDeviceToHost) );

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId) );
  printf("Device: %s\n", prop.name);

  float* Pt = (float*) malloc(Width * sizeof(float));
    
      //Assert Process
  char fileName[20] = "../matMul/matMul_";
  char bufferWidth[5] = " ";
  sprintf(bufferWidth, "%d", Width);
  strcat(fileName, bufferWidth);
  strcat(fileName, ".out");
  
  FILE *ptr_file;
  ptr_file =fopen(fileName, "r");

  assert(ptr_file); 
    
  for (int i=0; i < Width; i++){
    fscanf(ptr_file, "%f", &Pt[i]);
  }  

  fclose(ptr_file); 
   printf("Assertion started\n");
 for(int i=0 ;i<Width; i++) {
   assert(fabs(P[i * Width + i] - Pt[i]) < 0.1);
  }
      printf("Assertion Finished");

  // clean up memory
  free(M);
  free(N);
  free(P);
  free(Pt);
  checkCuda( hipFree(Md) );
  checkCuda( hipFree(Nd) );
  checkCuda( hipFree(Pd) );

  return 0;
}

